#include "hip/hip_runtime.h"
#include "kernel.h"
#include <hip/hip_runtime.h>
#include <>
#include <cmath>
#include <cstdio>
#include <cstring>

__global__ void stoch_rsi_kernel_batch(const GPUOHLCDataBatch_C *ohlc_batch, int num_symbols, int rsi_period, int stoch_period, GPUStochRSIResultBatch_C *results) {
    int symbol_idx = blockIdx.x;
    if (symbol_idx >= num_symbols) return;
    int count = ohlc_batch->counts[symbol_idx];
    if (rsi_period < 1) rsi_period = 1;
    int number_of_rsi = count - rsi_period + 1;
    if (number_of_rsi <= 0) {
        if (threadIdx.x == 0) {
            results->rsi[symbol_idx] = 50.0f;
            results->stoch_rsi_k[symbol_idx] = 50.0f;
            results->stoch_rsi_d[symbol_idx] = 50.0f;
        }
        return;
    }

    __shared__ float rsi_values[15];
    __shared__ float stoch_k[15];

    int idx = threadIdx.x;

    if (idx < number_of_rsi) {
        int k = rsi_period - 1 + idx;
        int start = k - rsi_period + 1;
        int end = k;

        float sum_gain = 0.0f;
        float sum_loss = 0.0f;

        if (rsi_period > 1) {
            for (int i = start + 1; i <= end; i++) {
                if (i < 15 && (i-1) < 15 && i >= 0 && (i-1) >= 0) {
                    float delta = ohlc_batch->close_prices[symbol_idx][i] - ohlc_batch->close_prices[symbol_idx][i - 1];
                    if (delta > 0) sum_gain += delta;
                    else if (delta < 0) sum_loss += -delta;
                }
            }
            float avg_gain = sum_gain / (float)(rsi_period - 1);
            float avg_loss = sum_loss / (float)(rsi_period - 1);
            float rs = (avg_loss > 0.000001f) ? (avg_gain / avg_loss) : (avg_gain > 0.000001f ? 1e10f : 1.0f);
            if (avg_loss == 0.0f && avg_gain == 0.0f) rs = 1.0f;
            rsi_values[idx] = 100.0f - (100.0f / (1.0f + rs));
        } else {
            rsi_values[idx] = 50.0f;
        }
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        if (stoch_period < 1) stoch_period = 1;

        for (int j = 0; j < number_of_rsi; j++) {
            int start_idx = fmaxf(0, j - stoch_period + 1);
            int end_idx = j;

            float min_rsi = rsi_values[start_idx];
            float max_rsi = rsi_values[start_idx];
            for (int m = start_idx; m <= end_idx; m++) {
                if (rsi_values[m] < min_rsi) min_rsi = rsi_values[m];
                if (rsi_values[m] > max_rsi) max_rsi = rsi_values[m];
            }

            float current_rsi = rsi_values[j];
            if (max_rsi > min_rsi) {
                stoch_k[j] = (current_rsi - min_rsi) / (max_rsi - min_rsi) * 100.0f;
            } else {
                stoch_k[j] = 50.0f;
            }
        }

        int last_rsi_idx = number_of_rsi - 1;
        if (last_rsi_idx >= 0) {
            results->rsi[symbol_idx] = rsi_values[last_rsi_idx];
            results->stoch_rsi_k[symbol_idx] = stoch_k[last_rsi_idx];

            int D_period = 3;
            float sum_k_for_d = 0.0f;
            int actual_d_count = 0;
            for (int p = 0; p < D_period; p++) {
                int k_idx_for_d = last_rsi_idx - p;
                if (k_idx_for_d >= 0) {
                    sum_k_for_d += stoch_k[k_idx_for_d];
                    actual_d_count++;
                } else {
                    break;
                }
            }
            if (actual_d_count > 0) {
                results->stoch_rsi_d[symbol_idx] = sum_k_for_d / (float)actual_d_count;
            } else {
                results->stoch_rsi_d[symbol_idx] = 50.0f;
            }
        } else {
            results->rsi[symbol_idx] = 50.0f;
            results->stoch_rsi_k[symbol_idx] = 50.0f;
            results->stoch_rsi_d[symbol_idx] = 50.0f;
        }
    }
}

__global__ void orderbook_kernel_batch(const GPUOrderBookDataBatch_C *orderbook_batch, int num_symbols, GPUOrderBookResultBatch_C *results) {
    int symbol_idx = blockIdx.x;
    if (symbol_idx >= num_symbols) return;

    __shared__ float partial_bid_sums[32];
    __shared__ float partial_ask_sums[32];

    int idx = threadIdx.x;
    int bid_count = orderbook_batch->bid_counts[symbol_idx];
    int ask_count = orderbook_batch->ask_counts[symbol_idx];

    if (idx < 10) {
        if (idx < bid_count) {
            partial_bid_sums[idx] = orderbook_batch->bid_quantities[symbol_idx][idx];
        } else {
            partial_bid_sums[idx] = 0.0f;
        }

        if (idx < ask_count) {
            partial_ask_sums[idx] = orderbook_batch->ask_quantities[symbol_idx][idx];
        } else {
            partial_ask_sums[idx] = 0.0f;
        }
    } else {
        if (idx < 32) {
            partial_bid_sums[idx] = 0.0f;
            partial_ask_sums[idx] = 0.0f;
        }
    }
    __syncthreads();

    for (int stride = 16; stride > 0; stride >>= 1) {
        if (idx < stride) {
            partial_bid_sums[idx] += partial_bid_sums[idx + stride];
            partial_ask_sums[idx] += partial_ask_sums[idx + stride];
        }
        __syncthreads();
    }

    if (idx == 0) {
        float total_bid = partial_bid_sums[0];
        float total_ask = partial_ask_sums[0];
        results->total_bid_volume[symbol_idx] = total_bid;
        results->total_ask_volume[symbol_idx] = total_ask;

        float total_volume = total_bid + total_ask;
        if (total_volume > 0.000001f) {
            results->bid_percentage[symbol_idx] = (total_bid / total_volume) * 100.0f;
            results->ask_percentage[symbol_idx] = (total_ask / total_volume) * 100.0f;
        } else {
            results->bid_percentage[symbol_idx] = 50.0f;
            results->ask_percentage[symbol_idx] = 50.0f;
        }
    }
}

static KernelError map_cuda_error(hipError_t cuda_err, const char* context) {
    if (cuda_err == hipSuccess) {
        return KERNEL_SUCCESS;
    }
    static char error_msg[256];
    snprintf(error_msg, sizeof(error_msg), "%s: %s", context, hipGetErrorString(cuda_err));
    return { cuda_err, error_msg };
}

static KernelError launch_stoch_rsi_kernel_internal(
    const GPUOHLCDataBatch_C *d_ohlc_batch,
    GPUStochRSIResultBatch_C *d_results,
    int num_symbols,
    int rsi_period,
    int stoch_period)
{
    const int THREADS_PER_BLOCK = 32;
    if (num_symbols > 0) {
        stoch_rsi_kernel_batch<<<num_symbols, THREADS_PER_BLOCK>>>(d_ohlc_batch, num_symbols, rsi_period, stoch_period, d_results);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA StochRSI kernel launch failed");
        }
    }
    return KERNEL_SUCCESS;
}

static KernelError launch_orderbook_kernel_internal(
    const GPUOrderBookDataBatch_C *d_orderbook_batch,
    GPUOrderBookResultBatch_C *d_results,
    int num_symbols)
{
    const int THREADS_PER_BLOCK = 32;
    if (num_symbols > 0) {
        orderbook_kernel_batch<<<num_symbols, THREADS_PER_BLOCK>>>(d_orderbook_batch, num_symbols, d_results);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Orderbook kernel launch failed");
        }
    }
    return KERNEL_SUCCESS;
}

extern "C" {
    KernelError cuda_wrapper_init_device(int device_id) {
        hipError_t err = hipSetDevice(device_id);
        if (err != hipSuccess) {
            return map_cuda_error(err, "Failed to set CUDA device");
        }
        return KERNEL_SUCCESS;
    }

    KernelError cuda_wrapper_reset_device() {
        hipError_t err = hipDeviceReset();
        if (err != hipSuccess) {
            return map_cuda_error(err, "Device reset failed");
        }
        return KERNEL_SUCCESS;
    }

    KernelError cuda_wrapper_get_device_count(int* count) {
        hipError_t err = hipGetDeviceCount(count);
        if (err != hipSuccess) {
            return map_cuda_error(err, "Failed to get device count");
        }
        return KERNEL_SUCCESS;
    }

    KernelError cuda_wrapper_get_device_properties(int device_id, struct hipDeviceProp_t* props) {
        hipError_t err = hipGetDeviceProperties(props, device_id);
        if (err != hipSuccess) {
            return map_cuda_error(err, "Failed to get device properties");
        }
        return KERNEL_SUCCESS;
    }

    KernelError cuda_wrapper_select_best_device(int* best_device_id_out) {
        int device_count = 0;
        hipError_t err = hipGetDeviceCount(&device_count);
        if (err != hipSuccess) {
            return map_cuda_error(err, "Failed to get device count");
        }
        if (device_count == 0) {
            return KERNEL_ERROR_NO_DEVICE;
        }

        int best_device = 0;
        int max_compute_capability = 0;

        for (int i = 0; i < device_count; i++) {
            hipDeviceProp_t props;
            err = hipGetDeviceProperties(&props, i);
            if (err == hipSuccess) {
                int current_compute_capability = props.major * 100 + props.minor;
                if (current_compute_capability > max_compute_capability) {
                    max_compute_capability = current_compute_capability;
                    best_device = i;
                }
            } else {
                return map_cuda_error(err, "Failed to get properties for device");
            }
        }
        *best_device_id_out = best_device;
        return KERNEL_SUCCESS;
    }

    KernelError cuda_wrapper_allocate_memory(
        GPUOHLCDataBatch_C** d_ohlc_batch,
        GPUOrderBookDataBatch_C** d_orderbook_batch,
        GPUStochRSIResultBatch_C** d_stoch_result,
        GPUOrderBookResultBatch_C** d_orderbook_result
    ) {
        hipError_t err;

        err = hipMalloc((void**)d_ohlc_batch, sizeof(GPUOHLCDataBatch_C));
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Malloc failed for d_ohlc_batch");
        }
        err = hipMemset(*d_ohlc_batch, 0, sizeof(GPUOHLCDataBatch_C));
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Memset failed for d_ohlc_batch");
        }

        err = hipMalloc((void**)d_orderbook_batch, sizeof(GPUOrderBookDataBatch_C));
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Malloc failed for d_orderbook_batch");
        }
        err = hipMemset(*d_orderbook_batch, 0, sizeof(GPUOrderBookDataBatch_C));
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Memset failed for d_orderbook_batch");
        }

        err = hipMalloc((void**)d_stoch_result, sizeof(GPUStochRSIResultBatch_C));
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Malloc failed for d_stoch_result");
        }
        err = hipMemset(*d_stoch_result, 0, sizeof(GPUStochRSIResultBatch_C));
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Memset failed for d_stoch_result");
        }

        err = hipMalloc((void**)d_orderbook_result, sizeof(GPUOrderBookResultBatch_C));
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Malloc failed for d_orderbook_result");
        }
        err = hipMemset(*d_orderbook_result, 0, sizeof(GPUOrderBookResultBatch_C));
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Memset failed for d_orderbook_result");
        }

        return KERNEL_SUCCESS;
    }

    KernelError cuda_wrapper_free_memory(
        GPUOHLCDataBatch_C* d_ohlc_batch,
        GPUOrderBookDataBatch_C* d_orderbook_batch,
        GPUStochRSIResultBatch_C* d_stoch_result,
        GPUOrderBookResultBatch_C* d_orderbook_result
    ) {
        KernelError last_err = KERNEL_SUCCESS;
        hipError_t current_err;

        if (d_ohlc_batch) {
            current_err = hipFree(d_ohlc_batch);
            if (current_err != hipSuccess) {
                last_err = map_cuda_error(current_err, "CUDA Free failed for d_ohlc_batch");
            }
        }
        if (d_orderbook_batch) {
            current_err = hipFree(d_orderbook_batch);
            if (current_err != hipSuccess && last_err.code == 0) {
                last_err = map_cuda_error(current_err, "CUDA Free failed for d_orderbook_batch");
            }
        }
        if (d_stoch_result) {
            current_err = hipFree(d_stoch_result);
            if (current_err != hipSuccess && last_err.code == 0) {
                last_err = map_cuda_error(current_err, "CUDA Free failed for d_stoch_result");
            }
        }
        if (d_orderbook_result) {
            current_err = hipFree(d_orderbook_result);
            if (current_err != hipSuccess && last_err.code == 0) {
                last_err = map_cuda_error(current_err, "CUDA Free failed for d_orderbook_result");
            }
        }
        return last_err;
    }

    KernelError cuda_wrapper_run_stoch_rsi_batch(
        GPUOHLCDataBatch_C* d_ohlc_batch_ptr,
        GPUStochRSIResultBatch_C* d_results_ptr,
        const GPUOHLCDataBatch_C* h_ohlc_batch,
        GPUStochRSIResultBatch_C* h_results,
        int num_symbols,
        int rsi_period,
        int stoch_period
    ) {
        if (num_symbols == 0) return KERNEL_SUCCESS;
        hipError_t err;

        err = hipMemcpy(d_ohlc_batch_ptr, h_ohlc_batch, sizeof(GPUOHLCDataBatch_C), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Memcpy H2D failed for StochRSI input");
        }

        KernelError kerr = launch_stoch_rsi_kernel_internal(d_ohlc_batch_ptr, d_results_ptr, num_symbols, rsi_period, stoch_period);
        if (kerr.code != 0) {
            return kerr;
        }

        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA StochRSI kernel execution failed");
        }

        err = hipMemcpy(h_results, d_results_ptr, sizeof(GPUStochRSIResultBatch_C), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Memcpy D2H failed for StochRSI results");
        }

        return KERNEL_SUCCESS;
    }

    KernelError cuda_wrapper_run_orderbook_batch(
        GPUOrderBookDataBatch_C* d_orderbook_batch_ptr,
        GPUOrderBookResultBatch_C* d_results_ptr,
        const GPUOrderBookDataBatch_C* h_orderbook_batch,
        GPUOrderBookResultBatch_C* h_results,
        int num_symbols
    ) {
        if (num_symbols == 0) return KERNEL_SUCCESS;
        hipError_t err;

        err = hipMemcpy(d_orderbook_batch_ptr, h_orderbook_batch, sizeof(GPUOrderBookDataBatch_C), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Memcpy H2D failed for Orderbook input");
        }

        KernelError kerr = launch_orderbook_kernel_internal(d_orderbook_batch_ptr, d_results_ptr, num_symbols);
        if (kerr.code != 0) {
            return kerr;
        }

        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Orderbook kernel execution failed");
        }

        err = hipMemcpy(h_results, d_results_ptr, sizeof(GPUOrderBookResultBatch_C), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Memcpy D2H failed for Orderbook results");
        }

        return KERNEL_SUCCESS;
    }
}
