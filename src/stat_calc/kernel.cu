#include "hip/hip_runtime.h"
#include "kernel.h"
#include <hip/hip_runtime.h>
#include <>
#include <cmath>
#include <cstdio>
#include <cstring>

__global__ void rsi_kernel_batch(const GPUOHLCDataBatch_C *ohlc_batch, int num_symbols, int rsi_period, GPURSIResultBatch_C *rsi_results) {
    int symbol_idx = blockIdx.x;
    if (symbol_idx >= num_symbols) return;
    
    int count = ohlc_batch->counts[symbol_idx];

    if (count != 15) {
        if (threadIdx.x == 0) {
            rsi_results->valid_rsi_count[symbol_idx] = 0;
        }
        return;
    }
    
    if (rsi_period < 1) rsi_period = 1;
    
    int number_of_rsi = count - rsi_period;
    if (number_of_rsi <= 0) {
        if (threadIdx.x == 0) {
            rsi_results->valid_rsi_count[symbol_idx] = 0;
        }
        return;
    }
    
    int idx = threadIdx.x;
    
    if (idx < number_of_rsi) {
        int current_pos = rsi_period + idx;
        
        float sum_gain = 0.0f;
        float sum_loss = 0.0f;
        int valid_changes = 0;
        
        for (int i = current_pos - rsi_period + 1; i <= current_pos; i++) {
            if (i > 0 && i < count && (i-1) >= 0) {
                float delta = ohlc_batch->close_prices[symbol_idx][i] - ohlc_batch->close_prices[symbol_idx][i - 1];
                if (delta > 0) {
                    sum_gain += delta;
                } else if (delta < 0) {
                    sum_loss += -delta;
                }
                valid_changes++;
            }
        }
        
        if (valid_changes > 0) {
            float avg_gain = sum_gain / (float)valid_changes;
            float avg_loss = sum_loss / (float)valid_changes;
            
            if (avg_loss > 0.000001f) {
                float rs = avg_gain / avg_loss;
                rsi_results->rsi_values[symbol_idx][idx] = 100.0f - (100.0f / (1.0f + rs));
            } else {
                rsi_results->rsi_values[symbol_idx][idx] = (avg_gain > 0.000001f) ? 100.0f : 50.0f;
            }
        } else {
            rsi_results->rsi_values[symbol_idx][idx] = 50.0f;
        }
    }
    
    __syncthreads();
    
    if (threadIdx.x == 0) {
        rsi_results->valid_rsi_count[symbol_idx] = number_of_rsi;
    }
}

// __global__ void stoch_rsi_kernel_batch(const GPURSIResultBatch_C *rsi_results, int num_symbols, int stoch_period, GPUStochRSIResultBatch_C *stoch_results) {
//     int symbol_idx = blockIdx.x;
//     if (symbol_idx >= num_symbols) return;
    
//     int valid_rsi_count = rsi_results->valid_rsi_count[symbol_idx];

//     if (valid_rsi_count <= 15) {
//         return;
//     }
    
//     if (stoch_period < 1) stoch_period = 1;
    
//     __shared__ float stoch_k[15];
//     int idx = threadIdx.x;
    
//     if (idx < valid_rsi_count) {
//         int start_idx = (idx >= stoch_period - 1) ? (idx - stoch_period + 1) : 0;
//         int end_idx = idx;
        
//         float min_rsi = rsi_results->rsi_values[symbol_idx][start_idx];
//         float max_rsi = rsi_results->rsi_values[symbol_idx][start_idx];
        
//         for (int m = start_idx; m <= end_idx; m++) {
//             if (m < valid_rsi_count) { 
//                 float current_rsi_val = rsi_results->rsi_values[symbol_idx][m];
//                 if (current_rsi_val < min_rsi) min_rsi = current_rsi_val;
//                 if (current_rsi_val > max_rsi) max_rsi = current_rsi_val;
//             }
//         }
        
//         float current_rsi = rsi_results->rsi_values[symbol_idx][idx];
//         if (fabsf(max_rsi - min_rsi) > 0.000001f) {
//             stoch_k[idx] = ((current_rsi - min_rsi) / (max_rsi - min_rsi)) * 100.0f;
//         } else {
//             return;
//         }
//     }
    
//     __syncthreads();

//     if (threadIdx.x == 0) {
//         int last_rsi_idx = valid_rsi_count - 1;
//         if (last_rsi_idx >= 0) {
//             stoch_results->rsi[symbol_idx] = rsi_results->rsi_values[symbol_idx][last_rsi_idx];
//             stoch_results->stoch_rsi_k[symbol_idx] = stoch_k[last_rsi_idx];
            
//             int D_period = 15;
//             float sum_k_for_d = 0.0f;
//             int actual_d_count = 0;
            
//             for (int p = 0; p < D_period && p <= last_rsi_idx; p++) {
//                 int k_idx_for_d = last_rsi_idx - p;
//                 if (k_idx_for_d >= 0) {
//                     sum_k_for_d += stoch_k[k_idx_for_d];
//                     actual_d_count++;
//                 }
//             }
            
//             if (actual_d_count > 0) {
//                 stoch_results->stoch_rsi_d[symbol_idx] = sum_k_for_d / (float)actual_d_count;
//             } else {
//                 return;
//             }
//         } else {
//             return;
//         }
//     }
// }

__global__ void orderbook_kernel_batch(const GPUOrderBookDataBatch_C *orderbook_batch, int num_symbols, GPUOrderBookResultBatch_C *results) {
    int symbol_idx = blockIdx.x;
    if (symbol_idx >= num_symbols) return;

    __shared__ float partial_bid_sums[32];
    __shared__ float partial_ask_sums[32];

    int idx = threadIdx.x;
    int bid_count = orderbook_batch->bid_counts[symbol_idx];
    int ask_count = orderbook_batch->ask_counts[symbol_idx];

    if (idx < MAX_ORDERBOOK_SIZE) {
        if (idx < bid_count) {
            partial_bid_sums[idx] = orderbook_batch->bid_quantities[symbol_idx][idx];
        } else {
            partial_bid_sums[idx] = 0.0f;
        }

        if (idx < ask_count) {
            partial_ask_sums[idx] = orderbook_batch->ask_quantities[symbol_idx][idx];
        } else {
            partial_ask_sums[idx] = 0.0f;
        }
    } else {
        if (idx < 32) {
            partial_bid_sums[idx] = 0.0f;
            partial_ask_sums[idx] = 0.0f;
        }
    }
    __syncthreads();

    for (int stride = 16; stride > 0; stride >>= 1) {
        if (idx < stride) {
            partial_bid_sums[idx] += partial_bid_sums[idx + stride];
            partial_ask_sums[idx] += partial_ask_sums[idx + stride];
        }
        __syncthreads();
    }


    if (idx == 0) {
        // --- Existing Calculations ---
        float total_bid = partial_bid_sums[0];
        float total_ask = partial_ask_sums[0];
        results->total_bid_volume[symbol_idx] = total_bid;
        results->total_ask_volume[symbol_idx] = total_ask;

        float total_volume = total_bid + total_ask;
        if (total_volume > 0.000001f) {
            results->bid_percentage[symbol_idx] = (total_bid / total_volume) * 100.0f;
            results->ask_percentage[symbol_idx] = (total_ask / total_volume) * 100.0f;
        } else {
            results->bid_percentage[symbol_idx] = 50.0f;
            results->ask_percentage[symbol_idx] = 50.0f;
        }

        // --- MODIFIED SPREAD & NEW BEST BID/ASK LOGIC ---
        if (bid_count > 0 && ask_count > 0) {
            // Get best bid/ask from the top of the book (index 0)
            float best_bid = orderbook_batch->bid_prices[symbol_idx][0];
            float best_ask = orderbook_batch->ask_prices[symbol_idx][0];
            
            // Populate the new fields in the results struct
            results->best_bid_price[symbol_idx] = best_bid;
            results->best_ask_price[symbol_idx] = best_ask;
            results->best_bid_qty[symbol_idx] = orderbook_batch->bid_quantities[symbol_idx][0];
            results->best_ask_qty[symbol_idx] = orderbook_batch->ask_quantities[symbol_idx][0];
            
            // Calculate spread
            if (best_bid > 0.000001f && best_ask > 0.000001f && best_ask > best_bid) {
                float mid_price = (best_bid + best_ask) / 2.0f;
                float spread = best_ask - best_bid;
                results->spread_percentage[symbol_idx] = (spread / mid_price) * 100.0f; // This is now a percentage
            } else {
                results->spread_percentage[symbol_idx] = 1000.0f; // Invalid prices, set a high spread
            }
        } else {
            // Handle cases with a one-sided or empty book
            results->spread_percentage[symbol_idx] = 1000.0f; // High spread
            results->best_bid_price[symbol_idx] = 0.0f;
            results->best_ask_price[symbol_idx] = 0.0f;
            results->best_bid_qty[symbol_idx] = 0.0f;
            results->best_ask_qty[symbol_idx] = 0.0f;
        }
    }
}

static KernelError map_cuda_error(hipError_t cuda_err, const char* context) {
    if (cuda_err == hipSuccess) {
        return KERNEL_SUCCESS;
    }
    static char error_msg[256];
    snprintf(error_msg, sizeof(error_msg), "%s: %s", context, hipGetErrorString(cuda_err));
    return { cuda_err, error_msg };
}

static KernelError launch_rsi_kernel_internal(
    const GPUOHLCDataBatch_C *d_ohlc_batch,
    GPURSIResultBatch_C *d_rsi_results,
    int num_symbols,
    int rsi_period)
{
    const int THREADS_PER_BLOCK = 32;
    if (num_symbols > 0) {
        rsi_kernel_batch<<<num_symbols, THREADS_PER_BLOCK>>>(d_ohlc_batch, num_symbols, rsi_period, d_rsi_results);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA RSI kernel launch failed");
        }
    }
    return KERNEL_SUCCESS;
}

// static KernelError launch_stoch_rsi_kernel_internal(
//     const GPURSIResultBatch_C *d_rsi_results,
//     GPUStochRSIResultBatch_C *d_stoch_results,
//     int num_symbols,
//     int stoch_period)
// {
//     const int THREADS_PER_BLOCK = 32;
//     if (num_symbols > 0) {
//         stoch_rsi_kernel_batch<<<num_symbols, THREADS_PER_BLOCK>>>(d_rsi_results, num_symbols, stoch_period, d_stoch_results);
//         hipError_t err = hipGetLastError();
//         if (err != hipSuccess) {
//             return map_cuda_error(err, "CUDA StochRSI kernel launch failed");
//         }
//     }
//     return KERNEL_SUCCESS;
// }

static KernelError launch_orderbook_kernel_internal(
    const GPUOrderBookDataBatch_C *d_orderbook_batch,
    GPUOrderBookResultBatch_C *d_results,
    int num_symbols)
{
    const int THREADS_PER_BLOCK = 32;
    if (num_symbols > 0) {
        orderbook_kernel_batch<<<num_symbols, THREADS_PER_BLOCK>>>(d_orderbook_batch, num_symbols, d_results);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Orderbook kernel launch failed");
        }
    }
    return KERNEL_SUCCESS;
}

extern "C" {
    KernelError cuda_wrapper_init_device(int device_id) {
        hipError_t err = hipSetDevice(device_id);
        if (err != hipSuccess) {
            return map_cuda_error(err, "Failed to set CUDA device");
        }
        return KERNEL_SUCCESS;
    }
    
    KernelError cuda_wrapper_reset_device() {
        hipError_t err = hipDeviceReset();
        if (err != hipSuccess) {
            return map_cuda_error(err, "Device reset failed");
        }
        return KERNEL_SUCCESS;
    }
    
    KernelError cuda_wrapper_get_device_count(int* count) {
        hipError_t err = hipGetDeviceCount(count);
        if (err != hipSuccess) {
            return map_cuda_error(err, "Failed to get device count");
        }
        return KERNEL_SUCCESS;
    }
    
    KernelError cuda_wrapper_get_device_properties(int device_id, struct hipDeviceProp_t* props) {
        hipError_t err = hipGetDeviceProperties(props, device_id);
        if (err != hipSuccess) {
            return map_cuda_error(err, "Failed to get device properties");
        }
        return KERNEL_SUCCESS;
    }
    
    KernelError cuda_wrapper_select_best_device(int* best_device_id_out) {
        int device_count = 0;
        hipError_t err = hipGetDeviceCount(&device_count);
        if (err != hipSuccess) {
            return map_cuda_error(err, "Failed to get device count");
        }
        if (device_count == 0) {
            return KERNEL_ERROR_NO_DEVICE;
        }
        
        int best_device = 0;
        int max_compute_capability = 0;
        for (int i = 0; i < device_count; i++) {
            hipDeviceProp_t props;
            err = hipGetDeviceProperties(&props, i);
            if (err == hipSuccess) {
                int current_compute_capability = props.major * 100 + props.minor;
                if (current_compute_capability > max_compute_capability) {
                    max_compute_capability = current_compute_capability;
                    best_device = i;
                }
            } else {
                return map_cuda_error(err, "Failed to get properties for device");
            }
        }
        *best_device_id_out = best_device;
        return KERNEL_SUCCESS;
    }
    
    KernelError cuda_wrapper_allocate_memory(
        GPUOHLCDataBatch_C **d_ohlc_batch,
        GPUOrderBookDataBatch_C **d_orderbook_batch,
        GPURSIResultBatch_C **d_rsi_result,
        //GPUStochRSIResultBatch_C** d_stoch_result,
        GPUOrderBookResultBatch_C **d_orderbook_result
    ) {
        hipError_t err;
        
        err = hipMalloc((void**)d_ohlc_batch, sizeof(GPUOHLCDataBatch_C));
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Malloc failed for d_ohlc_batch");
        }
        err = hipMemset(*d_ohlc_batch, 0, sizeof(GPUOHLCDataBatch_C));
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Memset failed for d_ohlc_batch");
        }
        
        err = hipMalloc((void**)d_orderbook_batch, sizeof(GPUOrderBookDataBatch_C));
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Malloc failed for d_orderbook_batch");
        }
        err = hipMemset(*d_orderbook_batch, 0, sizeof(GPUOrderBookDataBatch_C));
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Memset failed for d_orderbook_batch");
        }
        
        err = hipMalloc((void**)d_rsi_result, sizeof(GPURSIResultBatch_C));
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Malloc failed for d_rsi_result");
        }
        err = hipMemset(*d_rsi_result, 0, sizeof(GPURSIResultBatch_C));
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Memset failed for d_rsi_result");
        }
        
        // err = hipMalloc((void**)d_stoch_result, sizeof(GPUStochRSIResultBatch_C));
        // if (err != hipSuccess) {
        //     return map_cuda_error(err, "CUDA Malloc failed for d_stoch_result");
        // }
        // err = hipMemset(*d_stoch_result, 0, sizeof(GPUStochRSIResultBatch_C));
        // if (err != hipSuccess) {
        //     return map_cuda_error(err, "CUDA Memset failed for d_stoch_result");
        // }
        
        err = hipMalloc((void**)d_orderbook_result, sizeof(GPUOrderBookResultBatch_C));
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Malloc failed for d_orderbook_result");
        }
        err = hipMemset(*d_orderbook_result, 0, sizeof(GPUOrderBookResultBatch_C));
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Memset failed for d_orderbook_result");
        }
        
        return KERNEL_SUCCESS;
    }
    
    KernelError cuda_wrapper_free_memory(
        GPUOHLCDataBatch_C *d_ohlc_batch,
        GPUOrderBookDataBatch_C *d_orderbook_batch,
        GPURSIResultBatch_C *d_rsi_result,
        //GPUStochRSIResultBatch_C* d_stoch_result,
        GPUOrderBookResultBatch_C *d_orderbook_result
    ) {
        KernelError last_err = KERNEL_SUCCESS;
        hipError_t current_err;
        
        if (d_ohlc_batch) {
            current_err = hipFree(d_ohlc_batch);
            if (current_err != hipSuccess) {
                last_err = map_cuda_error(current_err, "CUDA Free failed for d_ohlc_batch");
            }
        }
        
        if (d_orderbook_batch) {
            current_err = hipFree(d_orderbook_batch);
            if (current_err != hipSuccess && last_err.code == 0) {
                last_err = map_cuda_error(current_err, "CUDA Free failed for d_orderbook_batch");
            }
        }
        
        if (d_rsi_result) {
            current_err = hipFree(d_rsi_result);
            if (current_err != hipSuccess && last_err.code == 0) {
                last_err = map_cuda_error(current_err, "CUDA Free failed for d_rsi_result");
            }
        }
        
        // if (d_stoch_result) {
        //     current_err = hipFree(d_stoch_result);
        //     if (current_err != hipSuccess && last_err.code == 0) {
        //         last_err = map_cuda_error(current_err, "CUDA Free failed for d_stoch_result");
        //     }
        // }
        
        if (d_orderbook_result) {
            current_err = hipFree(d_orderbook_result);
            if (current_err != hipSuccess && last_err.code == 0) {
                last_err = map_cuda_error(current_err, "CUDA Free failed for d_orderbook_result");
            }
        }
        
        return last_err;
    }
    
    KernelError cuda_wrapper_run_rsi_batch(
        GPUOHLCDataBatch_C *d_ohlc_batch_ptr,
        GPURSIResultBatch_C *d_rsi_results_ptr,
        const GPUOHLCDataBatch_C *h_ohlc_batch,
        GPURSIResultBatch_C *h_rsi_results,
        int num_symbols,
        int rsi_period
    ) {
        if (num_symbols == 0) return KERNEL_SUCCESS;
        
        hipError_t err;
        
        err = hipMemcpy(d_ohlc_batch_ptr, h_ohlc_batch, sizeof(GPUOHLCDataBatch_C), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Memcpy H2D failed for RSI input");
        }
        
        KernelError kerr = launch_rsi_kernel_internal(d_ohlc_batch_ptr, d_rsi_results_ptr, num_symbols, rsi_period);
        if (kerr.code != 0) {
            return kerr;
        }
        
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA RSI kernel execution failed");
        }
        
        err = hipMemcpy(h_rsi_results, d_rsi_results_ptr, sizeof(GPURSIResultBatch_C), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Memcpy D2H failed for RSI results");
        }
        
        return KERNEL_SUCCESS;
    }
    
    // KernelError cuda_wrapper_run_stoch_rsi_batch(
    //     GPURSIResultBatch_C* d_rsi_results_ptr,
    //     GPUStochRSIResultBatch_C* d_stoch_results_ptr,
    //     const GPURSIResultBatch_C* h_rsi_results,
    //     GPUStochRSIResultBatch_C* h_stoch_results,
    //     int num_symbols,
    //     int stoch_period
    // ) {
    //     if (num_symbols == 0) return KERNEL_SUCCESS;
        
    //     hipError_t err;
        
    //     err = hipMemcpy(d_rsi_results_ptr, h_rsi_results, sizeof(GPURSIResultBatch_C), hipMemcpyHostToDevice);
    //     if (err != hipSuccess) {
    //         return map_cuda_error(err, "CUDA Memcpy H2D failed for StochRSI input");
    //     }
        
    //     KernelError kerr = launch_stoch_rsi_kernel_internal(d_rsi_results_ptr, d_stoch_results_ptr, num_symbols, stoch_period);
    //     if (kerr.code != 0) {
    //         return kerr;
    //     }
        
    //     err = hipDeviceSynchronize();
    //     if (err != hipSuccess) {
    //         return map_cuda_error(err, "CUDA StochRSI kernel execution failed");
    //     }
        
    //     err = hipMemcpy(h_stoch_results, d_stoch_results_ptr, sizeof(GPUStochRSIResultBatch_C), hipMemcpyDeviceToHost);
    //     if (err != hipSuccess) {
    //         return map_cuda_error(err, "CUDA Memcpy D2H failed for StochRSI results");
    //     }
        
    //     return KERNEL_SUCCESS;
    // }
    
    KernelError cuda_wrapper_run_orderbook_batch(
        GPUOrderBookDataBatch_C *d_orderbook_batch_ptr,
        GPUOrderBookResultBatch_C *d_results_ptr,
        const GPUOrderBookDataBatch_C *h_orderbook_batch,
        GPUOrderBookResultBatch_C *h_results,
        int num_symbols
    ) {
        if (num_symbols == 0) return KERNEL_SUCCESS;
        
        hipError_t err;
        
        err = hipMemcpy(d_orderbook_batch_ptr, h_orderbook_batch, sizeof(GPUOrderBookDataBatch_C), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Memcpy H2D failed for Orderbook input");
        }
        
        KernelError kerr = launch_orderbook_kernel_internal(d_orderbook_batch_ptr, d_results_ptr, num_symbols);
        if (kerr.code != 0) {
            return kerr;
        }
        
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Orderbook kernel execution failed");
        }
        
        err = hipMemcpy(h_results, d_results_ptr, sizeof(GPUOrderBookResultBatch_C), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            return map_cuda_error(err, "CUDA Memcpy D2H failed for Orderbook results");
        }
        
        return KERNEL_SUCCESS;
    }
    
    KernelError cuda_wrapper_get_device_info(int device_id, DeviceInfo* info) {
        hipDeviceProp_t prop;
        hipError_t err = hipGetDeviceProperties(&prop, device_id);
        if (err != hipSuccess) {
            return (KernelError){ .code = 10, .message = "Failed to get device properties" };
        }
        
        strncpy(info->name, prop.name, 256);
        info->major = prop.major;
        info->minor = prop.minor;
        info->totalGlobalMem = prop.totalGlobalMem;
        return (KernelError){ .code = 0, .message = "Success" };
    }
}
